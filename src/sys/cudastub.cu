/****************************************************************************
** 
**  Copyright (C) 2019-2020 Boris Krasnopolsky, Alexey Medvedev
**  Contact: xamg-test@imec.msu.ru
** 
**  This file is part of the XAMG library.
** 
**  Commercial License Usage
**  Licensees holding valid commercial XAMG licenses may use this file in
**  accordance with the terms of commercial license agreement.
**  The license terms and conditions are subject to mutual agreement
**  between Licensee and XAMG library authors signed by both parties
**  in a written form.
** 
**  GNU General Public License Usage
**  Alternatively, this file may be used under the terms of the GNU
**  General Public License, either version 3 of the License, or (at your
**  option) any later version. The license is as published by the Free 
**  Software Foundation and appearing in the file LICENSE.GPL3 included in
**  the packaging of this file. Please review the following information to
**  ensure the GNU General Public License requirements will be met:
**  https://www.gnu.org/licenses/gpl-3.0.html.
** 
****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <dlfcn.h>


extern "C" { 

void cuda_stub_setup() {
}
hipError_t CUDARTAPI hipConfigureCall(dim3 g, dim3 b, size_t sharedMem, hipStream_t stream)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}
hipError_t CUDARTAPI hipDeviceSynchronize(void)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}
hipError_t CUDARTAPI hipEventCreateWithFlags(hipEvent_t *event, unsigned int flags)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}
hipError_t CUDARTAPI hipEventDestroy(hipEvent_t event)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipEventRecord(hipEvent_t event, hipStream_t stream)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);    
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipFree(void *devPtr)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipHostFree(void *ptr)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t hipFuncGetAttributes(struct hipFuncAttributes *p, const void *c)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipGetDevice(int *device)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipGetDeviceCount(int *count)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipGetDeviceProperties(struct hipDeviceProp_t *prop, int device)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
const char* CUDARTAPI hipGetErrorString(hipError_t error)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return NULL;

}
hipError_t CUDARTAPI hipHostAlloc(void **pHost, size_t size, unsigned int flags)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipHostRegister(void *ptr, size_t size, unsigned int flags)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipHostUnregister(void *ptr)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipLaunchByPtr(const void *func)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipMalloc(void **devPtr, size_t size)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipMemcpyAsync(void *dst, const void *src, size_t count, enum hipMemcpyKind kind, hipStream_t stream)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipMemset(void *devPtr, int value, size_t count)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipSetDevice(int device)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);   
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipSetupArgument(const void *arg, size_t size, size_t offset)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipStreamCreateWithFlags(hipStream_t *pStream, unsigned int flags)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}
hipError_t CUDARTAPI hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;

}

hipError_t CUDARTAPI hipStreamDestroy(hipStream_t Stream)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}

void** CUDARTAPI  __cudaRegisterFatBinary(void *fatCubin)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return 0;

}
void CUDARTAPI __cudaRegisterFatBinaryEnd(void **)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return;
}
void CUDARTAPI __cudaUnregisterFatBinary(void **fatCubinHandle)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return;

}
void CUDARTAPI __cudaRegisterFunction(void   **fatCubinHandle, const char *hostFun, char *deviceFun, const char *deviceName, int thread_limit, uint3 *tid, uint3 *bid, dim3 *bDim, dim3 *gDim, int *wSize)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return;

}

char CUDARTAPI __cudaInitModule(void **fatCubinHandle) 
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return 0;
}

hipError_t CUDARTAPI hipEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end) 
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}

hipError_t CUDARTAPI hipEventSynchronize(hipEvent_t event) 
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}

hipError_t CUDARTAPI hipDeviceEnablePeerAccess(int peerDevice, unsigned int flags)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}

hipError_t CUDARTAPI hipDeviceCanAccessPeer(int* canAccessPeer, int device, int peerDevice)
{
#ifdef WITH_GPU_VERBOSE    
    printf(">> STUB: %s\n", __FUNCTION__);
#endif
    return hipSuccess;
}

}
