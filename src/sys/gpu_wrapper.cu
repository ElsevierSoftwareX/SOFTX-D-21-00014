#include "hip/hip_runtime.h"
/****************************************************************************
** 
**  Copyright (C) 2019-2020 Boris Krasnopolsky, Alexey Medvedev
**  Contact: xamg-test@imec.msu.ru
** 
**  This file is part of the XAMG library.
** 
**  Commercial License Usage
**  Licensees holding valid commercial XAMG licenses may use this file in
**  accordance with the terms of commercial license agreement.
**  The license terms and conditions are subject to mutual agreement
**  between Licensee and XAMG library authors signed by both parties
**  in a written form.
** 
**  GNU General Public License Usage
**  Alternatively, this file may be used under the terms of the GNU
**  General Public License, either version 3 of the License, or (at your
**  option) any later version. The license is as published by the Free 
**  Software Foundation and appearing in the file LICENSE.GPL3 included in
**  the packaging of this file. Please review the following information to
**  ensure the GNU General Public License requirements will be met:
**  https://www.gnu.org/licenses/gpl-3.0.html.
** 
****************************************************************************/

#include <string>
#include <map>
#include <vector>
#include <iostream>
#include "xamg_types.h"
#include "io/logout.h"


#define CUDA_CALL(X) { hipError_t err = X; if (err != hipSuccess) { throw std::runtime_error(hipGetErrorString(err)); } }


namespace XAMG {
namespace CUDA {

static inline bool check_cc20(hipDeviceProp_t &prop, std::string &reason) {
    bool ret = prop.major >= 2;
    if (!ret)
        reason += " compute capability check FAILED;";
    return ret;
}

static inline bool check_overlap(hipDeviceProp_t &prop, std::string &reason) {
    bool ret = prop.deviceOverlap;
    if (!ret)
        reason += " overlap capability check FAILED;";
    return ret;

}
#if 0
static inline bool check_mode(hipDeviceProp_t &prop, std::string &reason) {
    bool ret = prop.computeMode;
    if (!ret)
        reason += " compute mode check FAILED;";
    return ret;

}
#endif
static inline bool check_warpsize(hipDeviceProp_t &prop, std::string &reason) {
    bool ret = prop.warpSize == 32;
    if (!ret)
        reason += " warp size check FAILED;";
    return ret;

}

struct device_traits {
    device_traits(int _id, const std::string _name, size_t _shared_mem, size_t _global_mem, size_t _mpc, bool _ecc,
                  size_t _clock, bool _uva) : 
                  id(_id), name(_name), shared_mem(_shared_mem), global_mem(_global_mem), mpc(_mpc), 
                  ecc(_ecc), clock(_clock), uva(_uva) {}
    int id;
    const std::string name;
    size_t shared_mem;
    size_t global_mem;
    size_t mpc;
    bool ecc;
    size_t clock;
    bool uva;
    std::vector<size_t> neighbours;
    bool operator==(device_traits &that) {
        bool equal = true;
        equal = equal && name == that.name;
        equal = equal && shared_mem == that.shared_mem;
        equal = equal && global_mem == that.global_mem;
        equal = equal && mpc == that.mpc;
        equal = equal && ecc == that.ecc;
        equal = equal && clock == that.clock;
        return equal;
    }
    bool operator!=(device_traits &that) {
        return !this->operator==(that);
    }
};

struct gpu_conf {
    std::vector<device_traits> devices;
    bool gpu_conf_done = false;
    bool p2p_already_enabled = false;
    uint64_t fingerprint = 0;
};

// FIXME global variable
gpu_conf *gpuconf = nullptr;
void gpu_conf_init();

int getnumgpus() {
    gpu_conf_init();
    if (!gpuconf->gpu_conf_done)
        return 0;
    return gpuconf->devices.size();
}

void setgpu(int i) {
    gpu_conf_init();
    if (!gpuconf->gpu_conf_done)
        return;
    int j = 0;
    for (auto &d : gpuconf->devices) {
        if (d.id == i)
            break;
        j++;
    }
    if (j == (int)gpuconf->devices.size()) {
        assert(false && "Device with given id is not configured");
        return;
    }
    CUDA_CALL(hipSetDevice(gpuconf->devices[j].id));
    // return gpuconf->devices.size();
}

void gpu_conf_init() {
    if (gpuconf) {
        return;
    }
    gpuconf = new gpu_conf;
    gpu_conf &conf = *gpuconf;
    auto &devices = conf.devices;
    int n = 0;
    CUDA_CALL(hipGetDeviceCount(&n));
    if (n == 0) {
        conf.gpu_conf_done = true;
        return;
    }
    for (int i = 0; i < n; i++) {
        hipDeviceProp_t prop;
        CUDA_CALL(hipGetDeviceProperties(&prop, i));
        bool device_is_ok = true;
        std::string reason;
        device_is_ok = device_is_ok && check_cc20(prop, reason);
        device_is_ok = device_is_ok && check_overlap(prop, reason);
        device_is_ok = device_is_ok && check_warpsize(prop, reason);        
        if (device_is_ok) {
            bool device_is_uva = true;
            device_is_uva = device_is_uva && prop.unifiedAddressing;
            // for windows: device_is_uva = device_is_uva && prop.tccDriver;
            devices.push_back(
                device_traits(i, prop.name, prop.sharedMemPerBlock, prop.totalGlobalMem, 
                              prop.multiProcessorCount, prop.ECCEnabled, prop.clockRate, 
                              device_is_uva));
        } else {
            XAMG::out << XAMG::WARN << "GPU device dropped: name: " << prop.name << ":" << reason << std::endl;
        }
    }
    bool uniform = true;
    for (auto it = devices.begin(); it != devices.end(); ++it) {
        auto it2 = std::next(it);
        if (it2 != devices.end()) {
            if (*it != *it2) {
                uniform = false;
            }
        }
    }
    if (!uniform) {
        XAMG::out << XAMG::WARN << "gpuconf: GPUs in a system are different, non-uniform conf is not supported";
        conf.gpu_conf_done = true;
        return;
    }
    for (auto it = devices.begin(); it != devices.end(); ++it) {
        auto &dev = *it;
        if (!dev.uva)
            continue;
        CUDA_CALL(hipSetDevice(dev.id));
        for (auto it2 = std::next(it); it2 != devices.end(); ++it2) {
            auto &dev2 = *it2;
            if (!dev2.uva)
                continue;
            int flag1 = 0, flag2 = 0;            
            CUDA_CALL(hipDeviceCanAccessPeer(&flag1, dev.id, dev2.id));
            CUDA_CALL(hipDeviceCanAccessPeer(&flag2, dev2.id, dev.id));
            dev.neighbours.push_back(it2 - devices.begin());
            dev2.neighbours.push_back(it - devices.begin());
            if (!conf.p2p_already_enabled) {
                CUDA_CALL(hipDeviceEnablePeerAccess(dev2.id, 0));
            }
        }
    }
    conf.p2p_already_enabled = true;
    conf.gpu_conf_done = true;
}

}
}
